#include "hip/hip_runtime.h"
#include <gstream/cuda/datatype/device_slotted_page.h>
#include <gstream/datatype/pagedb.h>
#include <hip/hip_runtime.h>

// Define meta parameter for page type 
using vertex_id_t = uint8_t;
using page_id_t = uint8_t;
using record_offset_t = uint8_t;
using slot_offset_t = uint8_t;
using record_size_t = uint8_t;
using edge_payload_t = uint8_t;
using vertex_payload_t = uint8_t;
constexpr std::size_t PageSize = 64;

// Define page type
using page_t = gstream::slotted_page<vertex_id_t, page_id_t, record_offset_t, slot_offset_t, record_size_t, PageSize, edge_payload_t, vertex_payload_t>;
// Define device page type
using device_page_t = gstream::device_slotted_page_t<page_t>;

__global__ void kernel(device_page_t* pages, std::size_t count)
{
    for (std::size_t i = 0; i < count; ++i) {
        printf("@ page[%llu]--------------------------------------\n", i);
        printf("page type: %s\n",
            (pages[i].is_sp()) ?
               "small page" : (pages[i].is_lp_head()) ?
               "large page (head)" : "large page (extended)");
        
        std::uint64_t number_of_slots = pages[i].number_of_slots();
        printf("number of slots in the page: %llu\n", number_of_slots);
        for (std::size_t j = 0; j < number_of_slots; ++j)
            printf("- slot[%llu]\tVID: %u\tRec-OFF: %u\tV-PL: %u\n", 
                   j,
                   pages[i].slot(j).vertex_id,
                   pages[i].slot(j).record_offset,
                   pages[i].slot(j).vertex_payload );
        printf("\n");
    }
}

int main()
{
    // Read pages from file using container as vector
    auto pages = gstream::read_pages<page_t, std::vector>("wewv.pages");
    
    // Initialize: Allocate device buffer
    void* devbuf;
    hipMalloc(&devbuf, sizeof(page_t) * pages.size());
    // Copy host pages to device buffer
    hipMemcpy(devbuf, pages.data(), sizeof(page_t) * pages.size(), hipMemcpyHostToDevice);
    // Call kernel
    kernel <<< 1, 1 >>>(reinterpret_cast<device_page_t*>(devbuf), pages.size());
    
    // Finalize resources
    hipFree(devbuf);
    hipDeviceSynchronize();
    hipDeviceReset();

	return 0;
}